#include "hip/hip_runtime.h"
/**
 *   @brief  Starter Kit project for CUDA- and OpenMP-accelerated Python libraries.
 *   @author Ash Vardanian
 *   @date   August 10, 2024
 *   @file   cupy_starter.cu
 *   @see    https://github.com/ashvardanian/cuda-python-starter-kit
 */
#include <csignal>   // `std::signal`
#include <cstdint>   // `std::uint32_t`
#include <cstdio>    // `std::printf`
#include <cstdlib>   // `std::rand`
#include <cstring>   // `std::memset`
#include <stdexcept> // `std::runtime_error`
#include <thread>    // `std::thread::hardware_concurrency()`

/*
 *  Include the SIMD intrinsics for the target architecture.
 *  Arm: https://developer.arm.com/architectures/instruction-sets/intrinsics
 *  x86: https://www.intel.com/content/www/us/en/docs/intrinsics-guide
 */
#if defined(__ARM_NEON) || defined(__aarch64__)
#include <arm_neon.h>
#endif
#if defined(__AVX2__) || defined(__AVX512F__)
#include <immintrin.h>
#endif

/*  It's a good idea to specialize kernels for different architectures of GPUs.
 *  - Pascal (6.0) introduced half-precision.
 *  - Volta (7.0) introduced tensor cores.
 *  - Ampere (8.0) introduced TF32.
 *  - Hopper (9.0) introduced FP8. and integer SIMD instructions.
 */
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 300
#define CUPY_STARTER_KEPLER 1
#endif
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 600
#define CUPY_STARTER_PASCAL 1
#endif
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
#define CUPY_STARTER_VOLTA 1
#endif
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 890
#define CUPY_STARTER_AMPERE 1
#endif
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
#define CUPY_STARTER_HOPPER 1
#endif

#if defined(__NVCC__)
#include <hip/hip_runtime.h>         // `CUtensorMap`
#include <cudaTypedefs.h> // `PFN_cuTensorMapEncodeTiled`
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#if defined(CUPY_STARTER_VOLTA)
#include <cuda/barrier>
#endif
#endif

/*
 *  If we are only testing the raw kernels, we don't need to link to PyBind.
 *  That accelerates the build process and simplifies the configs.
 */
#if !defined(CUPY_STARTER_TEST)
#include <pybind11/numpy.h> // `array_t`
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

namespace py = pybind11;
#endif

using cell_idx_t = std::uint32_t;

enum class backend_t {
    openmp_k,
    cuda_k,
};

/**
 *  @brief Stores the interrupt signal status.
 */
volatile std::sig_atomic_t global_signal_status = 0;

static void signal_handler(int signal) { global_signal_status = signal; }

#pragma region OpenMP and CPU code

template <typename scalar_type>
using reduce_type = std::conditional_t< //
    std::is_floating_point_v<scalar_type>, double,
    std::conditional_t<std::is_signed_v<scalar_type>, std::int64_t, std::uint64_t>>;

template <typename scalar_type>
using matmul_type = std::conditional_t<                 //
    std::is_floating_point_v<scalar_type>, scalar_type, //
    std::conditional_t<std::is_signed_v<scalar_type>, std::int64_t, std::uint64_t>>;

/**
 *  @brief Performs a reduction operation on a 1D array using OpenMP for parallelization.
 *
 *  This function reduces a 1D array of elements of type `scalar_type` to a single value
 *  using an OpenMP-enabled parallel reduction. The reduction operation is performed
 *  using multiple threads to sum the elements of the array efficiently.
 *
 *  @tparam scalar_type The data type of the array elements (e.g., float, double).
 *
 *  @param data A pointer to the input array of elements of type `scalar_type`.
 *  @param length The number of elements in the input array.
 *
 *  @return reduce_type<scalar_type> The result of the reduction operation, which is the sum
 *  of all elements in the array.
 */
template <typename scalar_type>
reduce_type<scalar_type> openmp_reduce(scalar_type const* data, std::size_t length) noexcept {
    reduce_type<scalar_type> initial_value = 0;
#pragma omp parallel for reduction(+ : initial_value)
    for (cell_idx_t i = 0; i < length; i++)
        initial_value += data[i];
    // Should be same as `std::accumulate(data.data(), data.data() + length, initial_value)`
    return initial_value;
}

/**
 *  @brief Performs tiled matrix multiplication using OpenMP for parallelization.
 *
 *  This function computes the matrix product of two matrices A and B, storing the result
 *  in matrix C. The multiplication is performed using a tiled approach to optimize cache
 *  usage by copying tiles into stack-allocated arrays. The computation is parallelized
 *  using OpenMP to leverage multiple threads.
 *
 *  @tparam scalar_type The data type of the matrix elements (e.g., float, double).
 *  @tparam tile_size The size of the tiles used for blocking, defaulting to 16.
 *
 *  @param matrix_a Pointer to the input matrix A, stored in row-major order.
 *  @param matrix_b Pointer to the input matrix B, stored in row-major order.
 *  @param matrix_c Pointer to the output matrix C, stored in row-major order.
 *  @param num_rows_a The number of rows in matrix A.
 *  @param num_cols_b The number of columns in matrix B.
 *  @param num_cols_a The number of columns in matrix A, and the number of rows in matrix B.
 *  @param stride_a The stride (leading dimension) of matrix A.
 *  @param stride_b The stride (leading dimension) of matrix B.
 *  @param stride_c The stride (leading dimension) of matrix C.
 *
 *  This function performs the operation:
 *
 *    C = A * B
 *
 *  where A is a (num_rows_a x num_cols_a) matrix, B is a (num_cols_a x num_cols_b) matrix,
 *  and C is a (num_rows_a x num_cols_b) matrix. The computation is broken down into smaller
 *  tile-sized chunks, which are copied into stack-allocated arrays to improve cache efficiency.
 *  The workload is parallelized using OpenMP to distribute the computation across multiple threads.
 */
template <typename scalar_type, cell_idx_t tile_size = 16>                                        //
void openmp_matmul(                                                                               //
    scalar_type const* matrix_a, scalar_type const* matrix_b, matmul_type<scalar_type>* matrix_c, //
    cell_idx_t num_rows_a, cell_idx_t num_cols_b, cell_idx_t num_cols_a,                          //
    cell_idx_t stride_a, cell_idx_t stride_b, cell_idx_t stride_c) noexcept {

#pragma omp parallel for collapse(2)
    for (cell_idx_t i = 0; i < num_rows_a; i += tile_size) {
        for (cell_idx_t j = 0; j < num_cols_b; j += tile_size) {
            scalar_type local_tile_a[tile_size][tile_size];
            scalar_type local_tile_b[tile_size][tile_size];
            matmul_type<scalar_type> local_tile_c[tile_size][tile_size];

            // Initialize the local tile to zero
            std::memset(local_tile_c, 0, tile_size * tile_size * sizeof(matmul_type<scalar_type>));

            for (cell_idx_t k = 0; k < num_cols_a; k += tile_size) {
                // Load tiles into local memory
                for (cell_idx_t ii = 0; ii < tile_size; ++ii)
                    for (cell_idx_t kk = 0; kk < tile_size; ++kk)
                        local_tile_a[ii][kk] =                           //
                            (i + ii < num_rows_a && k + kk < num_cols_a) //
                                ? matrix_a[(i + ii) * stride_a + (k + kk)]
                                : 0;

                for (cell_idx_t kk = 0; kk < tile_size; ++kk)
                    for (cell_idx_t jj = 0; jj < tile_size; ++jj)
                        local_tile_b[kk][jj] =                           //
                            (k + kk < num_cols_a && j + jj < num_cols_b) //
                                ? matrix_b[(k + kk) * stride_b + (j + jj)]
                                : 0;

                // Perform multiplication on the local tiles
                for (cell_idx_t ii = 0; ii < tile_size; ++ii)
                    for (cell_idx_t jj = 0; jj < tile_size; ++jj)
                        for (cell_idx_t kk = 0; kk < tile_size; ++kk)
                            local_tile_c[ii][jj] +=
                                static_cast<matmul_type<scalar_type>>(local_tile_a[ii][kk]) * local_tile_b[kk][jj];
            }

            // Write the result back to the output matrix
            for (cell_idx_t ii = 0; ii < tile_size; ++ii)
                for (cell_idx_t jj = 0; jj < tile_size; ++jj)
                    if (i + ii < num_rows_a && j + jj < num_cols_b)
                        matrix_c[(i + ii) * stride_c + (j + jj)] = local_tile_c[ii][jj];
        }
    }
}

#pragma endregion OpenMP and CPU code

#pragma region CUDA

#if defined(__NVCC__)

/**
 *  @brief Performs a reduction operation on a 1D array using CUDA.
 *
 *  This function reduces a contiguous 1D array of elements of type `scalar_type`
 *  to a single value using a CUDA-enabled reduction operation. The reduction
 *  operation is performed on the GPU using the Thrust library, which efficiently
 *  computes the sum (or another binary operation) of all elements in the array.
 *
 *  @tparam scalar_type The data type of the array elements (e.g., float, double).
 *
 *  @param data A pointer to the input array of elements of type `scalar_type`.
 *  @param length The number of elements in the input array.
 *
 *  @return reduce_type<scalar_type> The result of the reduction operation. For
 *  floating-point types, this is typically the sum of all elements in the array.
 */
template <typename scalar_type>
reduce_type<scalar_type> cuda_reduce(scalar_type const* data, std::size_t length) noexcept(false) {
    reduce_type<scalar_type> initial_value = 0;
    thrust::device_vector<scalar_type> device_array(data, data + length);
    return thrust::reduce(thrust::device, device_array.begin(), device_array.end(), initial_value);
}

/**
 *  @brief CUDA kernel for matrix multiplication with support for strided matrices.
 *
 *  This kernel computes the matrix product of two matrices A and B, storing the result in matrix C.
 *  The multiplication is performed in tiles of size `tile_size` to take advantage of shared memory
 *  for optimizing memory access patterns.
 *
 *  @tparam scalar_type The data type of the matrix elements (e.g., float, double).
 *  @tparam tile_size The size of the tiles used for shared memory, defaulting to 16.
 *
 *  @param matrix_a Pointer to the input matrix A, stored in row-major order.
 *  @param matrix_b Pointer to the input matrix B, stored in row-major order.
 *  @param matrix_c Pointer to the output matrix C, stored in row-major order.
 *  @param num_rows_a The number of rows in matrix A.
 *  @param num_cols_b The number of columns in matrix B.
 *  @param num_cols_a The number of columns in matrix A, and the number of rows in matrix B.
 *  @param stride_a The stride (leading dimension) of matrix A.
 *  @param stride_b The stride (leading dimension) of matrix B.
 *  @param stride_c The stride (leading dimension) of matrix C.
 *
 *  This kernel performs the operation:
 *
 *      C = A * B
 *
 *  where A is a (num_rows_a x num_cols_a) matrix, B is a (num_cols_a x num_cols_b) matrix,
 *  and C is a (num_rows_a x num_cols_b) matrix. The computation is broken down into smaller
 *  tile-sized chunks, which are loaded into shared memory to reduce global memory access overhead.
 *
 *  Each thread block computes a tile of the output matrix C by iterating over the corresponding
 *  tiles of matrices A and B. The kernel ensures correct handling of matrix boundaries and
 *  supports strided matrices, where elements of a row are not necessarily contiguous in memory.
 */
template <typename scalar_type, cell_idx_t tile_size = 16>                                        //
__global__ void cuda_matmul_kernel(                                                               //
    scalar_type const* matrix_a, scalar_type const* matrix_b, matmul_type<scalar_type>* matrix_c, //
    cell_idx_t num_rows_a, cell_idx_t num_cols_b, cell_idx_t num_cols_a,                          //
    cell_idx_t stride_a, cell_idx_t stride_b, cell_idx_t stride_c) {

    // Allocate shared memory for matrix_a and matrix_b tiles
    __shared__ scalar_type tile_a[tile_size][tile_size];
    __shared__ scalar_type tile_b[tile_size][tile_size];

    // Calculate the row and column index for this thread in the output matrix matrix_c
    cell_idx_t row = blockIdx.y * tile_size + threadIdx.y;
    cell_idx_t col = blockIdx.x * tile_size + threadIdx.x;

    // Accumulate the result for matrix_c[row][col]
    matmul_type<scalar_type> cell_c = 0;

    // Loop over tiles of matrix_a and matrix_b that are multiplied together
    for (cell_idx_t t = 0; t < (num_cols_a + tile_size - 1) / tile_size; ++t) {

        // Load tiles of matrix_a and matrix_b into shared memory with boundary checks
        tile_a[threadIdx.y][threadIdx.x] = //
            (row < num_rows_a && t * tile_size + threadIdx.x < num_cols_a)
                ? matrix_a[row * stride_a + t * tile_size + threadIdx.x]
                : 0;
        tile_b[threadIdx.y][threadIdx.x] = //
            (col < num_cols_b && t * tile_size + threadIdx.y < num_cols_a)
                ? matrix_b[(t * tile_size + threadIdx.y) * stride_b + col]
                : 0;

        // Synchronize to ensure all data is loaded into shared memory
        __syncthreads();

#pragma unroll
        // Perform the multiplication and accumulate
        for (cell_idx_t k = 0; k < tile_size; ++k) {
            cell_c += static_cast<matmul_type<scalar_type>>(tile_a[threadIdx.y][k]) * tile_b[k][threadIdx.x];
        }

        // Synchronize to ensure all threads are done with the current tile
        __syncthreads();
    }

    // Write the result back to the output matrix matrix_c with boundary check
    if (row < num_rows_a && col < num_cols_b)
        matrix_c[row * stride_c + col] = cell_c;
}

#endif // defined(__NVCC__)

#pragma endregion CUDA

#pragma region Python bindings
#if !defined(CUPY_STARTER_TEST)

/**
 *  @brief  Router function, that unpacks Python buffers into C++ pointers and calls the appropriate
 *          backend for reductions, like `openmp_reduce` or `cuda_reduce_kernel`.
 */
template <backend_t backend_kind, typename scalar_type>
static py::object python_reduce_typed(py::buffer_info const& buf) noexcept(false) {
    if (buf.ndim != 1 || buf.strides[0] != sizeof(scalar_type))
        throw std::runtime_error("Input should be a contiguous 1D array");
    scalar_type const* ptr = reinterpret_cast<scalar_type const*>(buf.ptr);
    reduce_type<scalar_type> result;

    if constexpr (backend_kind == backend_t::openmp_k) {
        // Explicitly enable dynamic teams, as the amount of compute per thread is not uniform.
        result = openmp_reduce<scalar_type>(ptr, buf.size);
    } else if constexpr (backend_kind == backend_t::cuda_k) {
#if defined(__NVCC__)
        result = cuda_reduce<scalar_type>(ptr, buf.size);
#else
        throw std::runtime_error("CUDA backend not available");
#endif
    } else {
        throw std::runtime_error("Unsupported backend");
    }

    return py::cast(result);
}

/**
 *  @brief  Router function, used to dispatch the right type-specific pre-compiled kernel
 *          using runtime-only type information. Calls `python_reduce_typed`.
 */
template <backend_t backend_kind> static py::object python_reduce(py::array a) noexcept(false) {
    if (py::isinstance<py::array_t<float>>(a))
        return python_reduce_typed<backend_kind, float>(a.request());
    else if (py::isinstance<py::array_t<double>>(a))
        return python_reduce_typed<backend_kind, double>(a.request());
    else if (py::isinstance<py::array_t<std::int8_t>>(a))
        return python_reduce_typed<backend_kind, std::int8_t>(a.request());
    else if (py::isinstance<py::array_t<std::uint8_t>>(a))
        return python_reduce_typed<backend_kind, std::uint8_t>(a.request());
    else if (py::isinstance<py::array_t<std::int16_t>>(a))
        return python_reduce_typed<backend_kind, std::int16_t>(a.request());
    else if (py::isinstance<py::array_t<std::uint16_t>>(a))
        return python_reduce_typed<backend_kind, std::uint16_t>(a.request());
    else if (py::isinstance<py::array_t<std::int32_t>>(a))
        return python_reduce_typed<backend_kind, std::int32_t>(a.request());
    else if (py::isinstance<py::array_t<std::uint32_t>>(a))
        return python_reduce_typed<backend_kind, std::uint32_t>(a.request());
    else if (py::isinstance<py::array_t<std::int64_t>>(a))
        return python_reduce_typed<backend_kind, std::int64_t>(a.request());
    else if (py::isinstance<py::array_t<std::uint64_t>>(a))
        return python_reduce_typed<backend_kind, std::uint64_t>(a.request());

    throw std::runtime_error("Unsupported data type");
    return py::none();
}

/**
 *  @brief  Router function, that unpacks Python buffers into C++ pointers and calls the appropriate
 *          backend for matrix multiplication, like `openmp_matmul` or `cuda_matmul_kernel`.
 */
template <backend_t backend_kind, typename scalar_type>
static py::array python_matmul_typed(py::buffer_info const& buffer_a, py::buffer_info const& buffer_b,
                                     std::size_t tile_size) {

    if (buffer_a.ndim != 2 || buffer_b.ndim != 2)
        throw std::runtime_error("Both tensors must be rank-2");
    if (buffer_a.shape[1] != buffer_b.shape[0])
        throw std::runtime_error("Inner dimensions must match");
    auto ptr_a = reinterpret_cast<scalar_type const*>(buffer_a.ptr);
    auto ptr_b = reinterpret_cast<scalar_type const*>(buffer_b.ptr);
    auto num_rows_a = static_cast<cell_idx_t>(buffer_a.shape[0]);
    auto num_cols_a = static_cast<cell_idx_t>(buffer_a.shape[1]);
    auto num_cols_b = static_cast<cell_idx_t>(buffer_b.shape[1]);
    auto stride_a = static_cast<cell_idx_t>(buffer_a.strides[0] / sizeof(scalar_type));
    auto stride_b = static_cast<cell_idx_t>(buffer_b.strides[0] / sizeof(scalar_type));

    // Allocate NumPy array for the result
    auto tensor_c = py::array_t<matmul_type<scalar_type>>({num_rows_a, num_cols_b});
    auto buffer_c = tensor_c.request();
    auto ptr_c = reinterpret_cast<matmul_type<scalar_type>*>(buffer_c.ptr);
    auto stride_c = static_cast<cell_idx_t>(buffer_c.strides[0] / sizeof(matmul_type<scalar_type>));

    // Call the appropriate kernel based on the backend
    using kernel_t = void (*)(scalar_type const*, scalar_type const*, matmul_type<scalar_type>*, cell_idx_t, cell_idx_t,
                              cell_idx_t, cell_idx_t, cell_idx_t, cell_idx_t);

    if constexpr (backend_kind == backend_t::openmp_k) {
        // Explicitly disable dynamic teams, as the amount of compute per thread is uniform.
        kernel_t kernel = nullptr;
        switch (tile_size) {
        case 4: kernel = &openmp_matmul<scalar_type, 4>; break;
        case 8: kernel = &openmp_matmul<scalar_type, 8>; break;
        case 16: kernel = &openmp_matmul<scalar_type, 16>; break;
        case 32: kernel = &openmp_matmul<scalar_type, 32>; break;
        case 64: kernel = &openmp_matmul<scalar_type, 64>; break;
        default: throw std::runtime_error("Unsupported tile size - choose from 4, 8, 16, 32, and 64");
        }
        kernel(ptr_a, ptr_b, ptr_c, num_rows_a, num_cols_b, num_cols_a, stride_a, stride_b, stride_c);

    } else if constexpr (backend_kind == backend_t::cuda_k) {
#if defined(__NVCC__)

        // Now allocate enough managed memory for all 3 matrices, and asyncronously copy them to the GPU,
        // using the 2D `memcpy2DAsync` function, which is more efficient than `memcpy` for large matrices.
        //
        // https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY.html
        // Allocate pitched memory for matrices A and B to ensure proper row alignment
        size_t pitch_a, pitch_b;
        scalar_type *ptr_a_cuda = nullptr, *ptr_b_cuda = nullptr;
        matmul_type<scalar_type>* ptr_c_cuda = nullptr;
        hipError_t error;

        // Allocate pitched memory for matrices A and B to ensure proper row alignment
        error = hipMallocPitch(&ptr_a_cuda, &pitch_a, num_cols_a * sizeof(scalar_type), num_rows_a);
        if (error != hipSuccess)
            throw std::runtime_error("Failed to allocate pitched memory for matrix A");

        error = hipMallocPitch(&ptr_b_cuda, &pitch_b, num_cols_b * sizeof(scalar_type), num_cols_a);
        if (error != hipSuccess) {
            hipFree(ptr_a_cuda);
            throw std::runtime_error("Failed to allocate pitched memory for matrix B");
        }

        // Allocate memory for matrix C (no pitch needed)
        error = hipMalloc(&ptr_c_cuda, num_rows_a * num_cols_b * sizeof(matmul_type<scalar_type>));
        if (error != hipSuccess) {
            hipFree(ptr_a_cuda);
            hipFree(ptr_b_cuda);
            throw std::runtime_error("Failed to allocate memory for matrix C");
        }

        // Copy matrices A and B from host to device using pitched memory
        error = hipMemcpy2D(ptr_a_cuda, pitch_a, buffer_a.ptr, buffer_a.strides[0], num_cols_a * sizeof(scalar_type),
                             num_rows_a, hipMemcpyHostToDevice);
        if (error != hipSuccess) {
            hipFree(ptr_a_cuda);
            hipFree(ptr_b_cuda);
            hipFree(ptr_c_cuda);
            throw std::runtime_error("Failed to copy matrix A to device");
        }

        error = hipMemcpy2D(ptr_b_cuda, pitch_b, buffer_b.ptr, buffer_b.strides[0], num_cols_b * sizeof(scalar_type),
                             num_cols_a, hipMemcpyHostToDevice);
        if (error != hipSuccess) {
            hipFree(ptr_a_cuda);
            hipFree(ptr_b_cuda);
            hipFree(ptr_c_cuda);
            throw std::runtime_error("Failed to copy matrix B to device");
        }

        // Initialize the result matrix C (zero it out)
        error = hipMemset(ptr_c_cuda, 0, num_rows_a * num_cols_b * sizeof(matmul_type<scalar_type>));
        if (error != hipSuccess) {
            hipFree(ptr_a_cuda);
            hipFree(ptr_b_cuda);
            hipFree(ptr_c_cuda);
            throw std::runtime_error("Failed to zero out matrix C");
        }

        // Synchronize to ensure all CUDA operations (including memory copies) are complete
        error = hipDeviceSynchronize();
        if (error != hipSuccess) {
            hipFree(ptr_a_cuda);
            hipFree(ptr_b_cuda);
            hipFree(ptr_c_cuda);
            throw std::runtime_error("CUDA operations did not complete successfully");
        }

        dim3 block_size(tile_size, tile_size);
        dim3 grid_size((num_cols_b + tile_size - 1) / tile_size, (num_rows_a + tile_size - 1) / tile_size);

        // Launch the CUDA kernel
        kernel_t kernel = nullptr;
        switch (tile_size) {
        case 4: kernel = &cuda_matmul_kernel<scalar_type, 4>; break;
        case 8: kernel = &cuda_matmul_kernel<scalar_type, 8>; break;
        case 16: kernel = &cuda_matmul_kernel<scalar_type, 16>; break;
        case 32: kernel = &cuda_matmul_kernel<scalar_type, 32>; break;
        case 64: kernel = &cuda_matmul_kernel<scalar_type, 64>; break;
        default: throw std::runtime_error("Unsupported tile size - choose from 4, 8, 16, 32, and 64");
        }

        kernel<<<grid_size, block_size>>>(ptr_a_cuda, ptr_b_cuda, ptr_c_cuda, num_rows_a, num_cols_b, num_cols_a,
                                          pitch_a / sizeof(scalar_type), pitch_b / sizeof(scalar_type), num_cols_b);

        // Check for errors during kernel launch
        error = hipGetLastError();
        if (error != hipSuccess) {
            hipFree(ptr_a_cuda);
            hipFree(ptr_b_cuda);
            hipFree(ptr_c_cuda);
            throw std::runtime_error(hipGetErrorString(error));
        }

        // Synchronize to ensure kernel execution is complete
        error = hipDeviceSynchronize();
        if (error != hipSuccess) {
            hipFree(ptr_a_cuda);
            hipFree(ptr_b_cuda);
            hipFree(ptr_c_cuda);
            throw std::runtime_error("CUDA operations did not complete successfully");
        }

        // Copy data from the GPU to the NumPy array
        error = hipMemcpy(ptr_c, ptr_c_cuda, num_rows_a * num_cols_b * sizeof(matmul_type<scalar_type>),
                           hipMemcpyDeviceToHost);
        if (error != hipSuccess) {
            hipFree(ptr_a_cuda);
            hipFree(ptr_b_cuda);
            hipFree(ptr_c_cuda);
            throw std::runtime_error("Failed to copy data from device to host");
        }

        // Free the GPU memory
        hipFree(ptr_a_cuda);
        hipFree(ptr_b_cuda);
        hipFree(ptr_c_cuda);

#else
        throw std::runtime_error("CUDA backend not available");
#endif
    } else {
        throw std::runtime_error("Unsupported backend");
    }

    return tensor_c;
}

/**
 *  @brief  Router function, used to dispatch the right type-specific pre-compiled kernel
 *          using runtime-only type information. Calls `python_matmul_typed`.
 */
template <backend_t backend_kind>
static py::array python_matmul(py::array a, py::array b, std::size_t tile_size) noexcept(false) {

    if (py::isinstance<py::array_t<float>>(a))
        return python_matmul_typed<backend_kind, float>(a.request(), b.request(), tile_size);
    else if (py::isinstance<py::array_t<double>>(a))
        return python_matmul_typed<backend_kind, double>(a.request(), b.request(), tile_size);
    else if (py::isinstance<py::array_t<std::int8_t>>(a))
        return python_matmul_typed<backend_kind, std::int8_t>(a.request(), b.request(), tile_size);
    else if (py::isinstance<py::array_t<std::uint8_t>>(a))
        return python_matmul_typed<backend_kind, std::uint8_t>(a.request(), b.request(), tile_size);
    else if (py::isinstance<py::array_t<std::int16_t>>(a))
        return python_matmul_typed<backend_kind, std::int16_t>(a.request(), b.request(), tile_size);
    else if (py::isinstance<py::array_t<std::uint16_t>>(a))
        return python_matmul_typed<backend_kind, std::uint16_t>(a.request(), b.request(), tile_size);
    else if (py::isinstance<py::array_t<std::int32_t>>(a))
        return python_matmul_typed<backend_kind, std::int32_t>(a.request(), b.request(), tile_size);
    else if (py::isinstance<py::array_t<std::uint32_t>>(a))
        return python_matmul_typed<backend_kind, std::uint32_t>(a.request(), b.request(), tile_size);
    else if (py::isinstance<py::array_t<std::int64_t>>(a))
        return python_matmul_typed<backend_kind, std::int64_t>(a.request(), b.request(), tile_size);
    else if (py::isinstance<py::array_t<std::uint64_t>>(a))
        return python_matmul_typed<backend_kind, std::uint64_t>(a.request(), b.request(), tile_size);

    throw std::runtime_error("Unsupported data type");
    return py::none();
}

PYBIND11_MODULE(cupy_starter, m) {

    std::signal(SIGINT, signal_handler);

    m.def("supports_cuda", []() -> bool {
#if defined(__NVCC__)
        return true;
#else
        return false;
#endif
    });

    m.def("log_cuda_devices", []() {
#if defined(__NVCC__)
        int device_count;
        hipDeviceProp_t device_props;
        hipError_t error = hipGetDeviceCount(&device_count);
        if (error != hipSuccess)
            throw std::runtime_error("Failed to get device count");
        for (int i = 0; i < device_count; i++) {
            error = hipGetDeviceProperties(&device_props, i);
            if (error != hipSuccess)
                throw std::runtime_error("Failed to get device properties");
            std::printf("Device %d: %s\n", i, device_props.name);
            std::printf("\tSMs: %d\n", device_props.multiProcessorCount);
            std::printf("\tGlobal mem: %.2fGB\n",
                        static_cast<float>(device_props.totalGlobalMem) / (1024 * 1024 * 1024));
            std::printf("\tCUDA Cap: %d.%d\n", device_props.major, device_props.minor);
        }
#else
        throw std::runtime_error("No CUDA devices available\n");
#endif
    });

    // This is how we could have used `thrust::` for higher-level operations
    m.def("reduce_openmp", &python_reduce<backend_t::openmp_k>);
    m.def("matmul_openmp", &python_matmul<backend_t::openmp_k>, py::arg("a"), py::arg("b"), py::kw_only(),
          py::arg("tile_size") = 16);

    m.def("reduce_cuda", &python_reduce<backend_t::cuda_k>);
    m.def("matmul_cuda", &python_matmul<backend_t::cuda_k>, py::arg("a"), py::arg("b"), py::kw_only(),
          py::arg("tile_size") = 16);
}

#endif // !defined(CUPY_STARTER_TEST)
#pragma endregion Python bindings

#if defined(CUPY_STARTER_TEST)

#include <algorithm> // `std::generate`
#include <numeric>   // `std::accumulate`
#include <vector>    // `std::vector`

int main() {

    // As a test, let's generate some random floats and reduce them.
    constexpr std::size_t num_elements = 1 << 20;
    std::vector<float> data(num_elements);
    std::generate(data.begin(), data.end(), []() { return static_cast<float>(std::rand() % 100); });

    // Let's test the OpenMP reduction
    double result = openmp_reduce(data.data(), num_elements);
    std::printf("OpenMP reduction result: %.2f\n", result);
    reduce_type<float> expected = std::accumulate(data.begin(), data.end(), 0.0);
    if (std::abs(result - expected) > 1e-6)
        throw std::runtime_error("OpenMP reduction failed");

#if defined(__NVCC__)
    // Let's test the CUDA reduction
    reduce_type<float> result_cuda = cuda_reduce(data.data(), num_elements);
    std::printf("CUDA reduction result: %.2f\n", result_cuda);
    if (std::abs(result_cuda - expected) > 1e-2)
        throw std::runtime_error("CUDA reduction failed");
#endif

    // Let's test the OpenMP matrix multiplication against CUDA
    constexpr cell_idx_t num_rows = 256;
    constexpr cell_idx_t num_cols = 256;
    std::vector<float> matrix_a(num_rows * num_cols);
    std::vector<float> matrix_b(num_rows * num_cols);
    std::generate(matrix_a.begin(), matrix_a.end(), []() { return static_cast<float>(std::rand() % 100); });
    std::generate(matrix_b.begin(), matrix_b.end(), []() { return static_cast<float>(std::rand() % 100); });
    std::vector<matmul_type<float>> matrix_c(num_rows * num_cols);
    openmp_matmul(matrix_a.data(), matrix_b.data(), matrix_c.data(), num_rows, num_cols, num_cols, num_cols, num_cols,
                  num_cols);

#if defined(__NVCC__)
    constexpr cell_idx_t tile_size = 16;
    dim3 block_size(tile_size, tile_size);
    dim3 grid_size((num_rows + tile_size - 1) / tile_size, (num_cols + tile_size - 1) / tile_size);

    std::vector<matmul_type<float>> matrix_c_cuda(num_rows * num_cols);
    cuda_matmul_kernel<float, tile_size><<<grid_size, block_size>>>(matrix_a.data(), matrix_b.data(),
                                                                    matrix_c_cuda.data(), num_rows, num_cols, num_cols,
                                                                    num_cols, num_cols, num_cols);
    matmul_type<float> max_diff = 0;
    for (std::size_t i = 0; i < num_rows * num_cols; i++)
        max_diff = std::max<matmul_type<float>>(max_diff, std::abs(matrix_c[i] - matrix_c_cuda[i]));
    std::printf("Max difference between OpenMP and CUDA matmul: %.2f\n", max_diff);
    if (max_diff > 1e-2)
        throw std::runtime_error("Matmul kernels do not match");
#endif

    return 0;
}

#endif // defined(CUPY_STARTER_TEST)